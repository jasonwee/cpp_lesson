/**
* Simple Timer
* ------------
* Copyright (c) 2007-2008, Daniel Cederman and Philippas Tsigas
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following 
* conditions are met:
*
* Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer 
* in the documentation and/or other materials provided with the distribution.
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, 
* BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
* SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL 
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
* INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
* NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*
**/


#ifdef _MSC_VER

#include <windows.h>

class SimpleTimer
{
	LARGE_INTEGER starttime;
public:
	void start();
	double end(); 
};

void SimpleTimer::start()
{
	QueryPerformanceCounter(&starttime);
}

double SimpleTimer::end()
{
	LARGE_INTEGER endtime,freq;
	QueryPerformanceCounter(&endtime);
	QueryPerformanceFrequency(&freq);

	return ((double)(endtime.QuadPart-starttime.QuadPart))/((double)(freq.QuadPart/1000.0));
}


#else

#include <hip/hip_runtime.h>
#include <sys/time.h>

class SimpleTimer
{
	struct timeval starttime;
public:
	void start();
	double end(); 
};

void SimpleTimer::start()
{
	gettimeofday(&starttime,0);
}

double SimpleTimer::end()
{
	struct timeval endtime;
	gettimeofday(&endtime,0);

	return (endtime.tv_sec - starttime.tv_sec)*1000.0 + (endtime.tv_usec - starttime.tv_usec)/1000.0;
}


#endif
